#include "hip/hip_runtime.h"
#include "lock.cuh"

Lock::Lock()
{
	int state = 0;
	HANDLE_ERROR(hipMalloc((void**)&mutex, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(mutex, &state, sizeof(int), hipMemcpyHostToDevice));
}

Lock::~Lock()
{
	hipFree(mutex);
}

__device__ void Lock::lock()
{
	while (atomicCAS(mutex, 0, 1) != 0);
}

__device__ void Lock::unlock()
{
	atomicExch(mutex, 0);
}
