#pragma once

#include <stdlib.h>
#include "Logger.h"

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		LOG_ERROR("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(1);
	}
}

void CUDA_init()
{
	hipDeviceProp_t prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
}

